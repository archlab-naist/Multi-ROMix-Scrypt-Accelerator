#include "hip/hip_runtime.h"
// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include "scrypt.h"
#include <dirent.h>
#include <ctype.h>
#include <sys/time.h>

#define N 16384
#define M 1000000
#define MAXLOOP M/N
//#define N 6

#define checkCudaErrors(x) \
{ \
    hipGetLastError(); \
    x; \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) \
        printf("GPU: hipError_t %d (%s)\n", err, hipGetErrorString(err)); \
}


__device__ scrypt_cuda(SHA256_CTX *ctx, WORD *block, unsigned long block_len, int dklenP1, int N, int dklenP2, WORD* hash_out){
    hash_out = scrypt(ctx, block, block_len, dklenP1, dklenP2);
}

__global__ void scrypt_top_cuda(uint32_t max_loop) {

    uint32_t index  = blockIdx.x * blockDim.x + threadIdx.x;

    SHA256_CTX *ctx = new SHA256_CTX();
    char ver[]="20000000";
    char prev_block[]="48f4bdc6cbabf6e59d5714adc7caa1af293bc49c75d447c2fdc1843694d1ef56";
    char mrkl_root[]="f03a2314e267c0e67627a51aa8c7bcdd99a2d173deec41ab96945eb4c7e43dee";
    char time[9];
    char bits[9];
    little_endian(ver, sizeof(ver) - 1);
    little_endian(prev_block, sizeof(prev_block) - 1);
    little_endian(mrkl_root, sizeof(mrkl_root) - 1);
    
    // Get time

    struct tm t;
    time_t t_of_day;

    t.tm_year = 2019-1900;  // Year - 1900
    t.tm_mon = 3-1;           // Month, where 1 = jan
    t.tm_mday = 13;          // Day of the month
    t.tm_hour = 7+9;
    t.tm_min = 51;
    t.tm_sec = 51;
    t.tm_isdst = -1;        // Is DST on? 1 = yes, 0 = no, -1 = unknown
    t_of_day = mktime(&t);
    
    WORD *wtime = new WORD(t_of_day);
    endian_cvt(wtime);
    word_to_hex_eight(*wtime, time, 8);

    word_to_hex_eight(436330391, bits, 8); // bits -- input
    little_endian(bits, 8);
    char test_scrypt_in[153];


    
    int in_index = 0;
    WORD i;
    for( i = 0; i < sizeof(ver)-1; i++){
        test_scrypt_in[i]=ver[i];
    }
    in_index += sizeof(ver)-1;
    for( i = 0; i < sizeof(prev_block); i++){
        test_scrypt_in[in_index+i] = prev_block[i];
    }
    in_index += sizeof(prev_block)-1;
    for( i = 0; i < sizeof(mrkl_root); i++){
        test_scrypt_in[in_index+i] = mrkl_root[i];
    }
    in_index += sizeof(mrkl_root)-1;
    for( i = 0; i < sizeof(time); i++){
        test_scrypt_in[in_index+i] = time[i];
    }
    in_index += sizeof(time)-1;
    for( i = 0; i < sizeof(bits); i++){
        test_scrypt_in[in_index+i] = bits[i];
    }

    WORD *nonce = new WORD(data->i*THREAD_NO_NONCE);
    endian_cvt(nonce);
    WORD *test_scrypt_out_w = new WORD[8]();
    char *test_scrypt_out = new char[32*8](); 
    WORD test_scrypt_in_w[20];

    for (i = index*max_loop; i<(index+1)*max_loop; i++){
        hex_string_to_words(test_scrypt_in, sizeof(test_scrypt_in), test_scrypt_in_w);
        test_scrypt_in_w[19] = i;
        endian_cvt(&test_scrypt_in_w[19]);
        test_scrypt_out_w = scrypt_cuda(ctx, test_scrypt_in_w, 20, 256, 1024, 1024);
        if(i==(index+1)*max_loop-1){
            printf("\nThread id: %d, nonce: %d\n", data->i, *nonce);
        }
    }


}


int main(int argc))
{

    
	int GPU_N;
	checkCudaErrors(hipGetDeviceCount(&GPU_N));
	printf("CUDA-capable device count: %d\n", GPU_N);
	checkCudaErrors(hipSetDevice(GPU_N-1));




    uint32_t blockSize = 256;
    uint32_t numBlocks = (N + blockSize - 1) / blockSize;
    // uint32_t *max_loop_cpu = (uint32_t *)malloc(sizeof(uint32_t));
    // *max_loop_cpu = M;
	// checkCudaErrors(hipMallocManaged(&max_loop_gpu, sizeof(uint32_t)));
    // hipMemcpy(max_loop_gpu, max_loop_cpu, hipMemcpyHostToDevice);

	
    scrypt_cuda <<<numBlocks, blockSize>>> (MAXLOOP);

	hipDeviceReset();
	
	return argc - 1;
}	
	
