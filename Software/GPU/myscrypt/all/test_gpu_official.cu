#include <stdint.h>
#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <algorithm>
#include <stddef.h>
// helper functions and utilities to work with CUDA
// #include <helper_functions.h>
// #include <helper_cuda.h>
#define CORRECTNONCE 235472032
#define MAXNONCE 10000000
#define NUMTHREAD 65536
#define BLOCKSIZE 256
#define STARTATNONCE CORRECTNONCE-MAXNONCE+1

// -----------------------------------------------------------------------------------------------------------------------------------
typedef unsigned char BYTE; // 8-bit byte
typedef uint32_t  WORD; // 32-bit word
// -----------------------------------------------------------------------------------------------------------------------------------
#define ipad_elm 0x36363636
#define opad_elm 0x5c5c5c5c
#define SUM(a,b) (a+b) & 0xffffffff
// -----------------------------------------------------------------------------------------------------------------------------------
#define h0 0x6a09e667
#define h1 0xbb67ae85
#define h2 0x3c6ef372
#define h3 0xa54ff53a
#define h4 0x510e527f
#define h5 0x9b05688c
#define h6 0x1f83d9ab
#define h7 0x5be0cd19


// Macros
#define SHA256_BLOCK_SIZE 32 

#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))

#define CH(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x,y,z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x,2) ^ ROTRIGHT(x,13) ^ (ROTRIGHT(x,22)))
#define EP1(x) (ROTRIGHT(x,6) ^ ROTRIGHT(x,11) ^ (ROTRIGHT(x,25)))
#define SIG0(x) (ROTRIGHT(x,7) ^ ROTRIGHT(x,18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x,17) ^ ROTRIGHT(x,19) ^ ((x) >> 10))


typedef struct SHA256_CTX
{
    BYTE data[64];
    WORD datalen;
    unsigned long long bitlen;
    WORD state[8];
} SHA256_CTX;
typedef struct SHA256_CTX_W
{
    WORD data[16];
    WORD datalen;
    unsigned long long bitlen;
    WORD state[8];
} SHA256_CTX_W;



typedef struct SCRYPT_PKG{
    SHA256_CTX_W ctx;
    WORD input[20];
    WORD output[8];
    WORD mem[1024][32];
    WORD salt[21];
    WORD hmac_out[8];
    
    WORD in_ihash[37];
    WORD in_2_ihash[49];
    WORD pbkdf2_rm_out[32];
    WORD ihash[8];
    WORD khash[8];
    WORD in_ohash[24];
    
}SCRYPT_PKG;

// -----------------------------------------------------------------------------------------------------------------------------------

#define SALSA_MIX(destination ,a1, a2, b) (destination ^ (ROTLEFT(SUM(a1,a2),b)))






#define ipad_elm 0x36363636
#define opad_elm 0x5c5c5c5c
#define SUM(a,b) (a+b) & 0xffffffff


__device__ BYTE hex_char_to_byte(char hex_char){
    if(hex_char >= 'a' && hex_char <='f'){
        return hex_char - 'a' + 10;
    }
    else if(hex_char >='A' && hex_char <= 'F'){
        return hex_char - 'A' + 10;
    } else if (hex_char >='0' && hex_char <= '9')
    {
        return hex_char - '0';
    }
    return 0;
}

BYTE hex_char_to_byte_host(char hex_char){
    if(hex_char >= 'a' && hex_char <='f'){
        return hex_char - 'a' + 10;
    }
    else if(hex_char >='A' && hex_char <= 'F'){
        return hex_char - 'A' + 10;
    } else if (hex_char >='0' && hex_char <= '9')
    {
        return hex_char - '0';
    }
    return 0;
}
__device__ void hex_string_to_bytes(char hex_str_in[], unsigned long hex_str_len, BYTE bytes_out[]){
    for (int i = 0; i<hex_str_len-1; i+=2){
        bytes_out[i/2] = ((hex_char_to_byte(hex_str_in[i])) << 4) | (hex_char_to_byte(hex_str_in[i+1]));

    }
}

__device__ void hex_string_to_words(char hex_str_in[], unsigned long hex_str_len, WORD words_out[]){
    for (int i = 0; i<hex_str_len-1; i+=8){
        words_out[i/8] = (\
            hex_char_to_byte(hex_str_in[i])<<28|\
            (hex_char_to_byte(hex_str_in[i+1])<<24 & 0x0f000000)|\
            (hex_char_to_byte(hex_str_in[i+2])<<20 & 0x00f00000)|\
            (hex_char_to_byte(hex_str_in[i+3])<<16 & 0x000f0000)|\
            (hex_char_to_byte(hex_str_in[i+4])<<12 & 0x0000f000)|\
            (hex_char_to_byte(hex_str_in[i+5])<<8  & 0x00000f00)|\
            (hex_char_to_byte(hex_str_in[i+6])<<4  & 0x000000f0)|\
            (hex_char_to_byte(hex_str_in[i+7])     & 0x0000000f)\
        );
        // printf("%08x  %d\n", words_out[i/8], i/8);
    }
}

void hex_string_to_words_host(char hex_str_in[], unsigned long hex_str_len, WORD words_out[]){
    for (int i = 0; i<hex_str_len-1; i+=8){
        words_out[i/8] = (\
            hex_char_to_byte_host(hex_str_in[i])<<28|\
            (hex_char_to_byte_host(hex_str_in[i+1])<<24 & 0x0f000000)|\
            (hex_char_to_byte_host(hex_str_in[i+2])<<20 & 0x00f00000)|\
            (hex_char_to_byte_host(hex_str_in[i+3])<<16 & 0x000f0000)|\
            (hex_char_to_byte_host(hex_str_in[i+4])<<12 & 0x0000f000)|\
            (hex_char_to_byte_host(hex_str_in[i+5])<<8  & 0x00000f00)|\
            (hex_char_to_byte_host(hex_str_in[i+6])<<4  & 0x000000f0)|\
            (hex_char_to_byte_host(hex_str_in[i+7])     & 0x0000000f)\
        );
        // printf("%08x  %d\n", words_out[i/8], i/8);
    }
}
__device__ void half_byte_to_hex(BYTE half_byte_in, char *hex){
    BYTE half_byte_conv = half_byte_in & 0x0f;
    if(half_byte_conv<16){
        if (half_byte_conv>=10){
            *hex = 'a'+ half_byte_conv - 10;
            // printf("%c\n", *hex);
            return;
        }
        else{
            *hex = '0' + half_byte_conv;
            // printf("%c\n", *hex);
            return;
        }
    }
    printf("The half byte must be in range of [0:15]\n");
}

void half_byte_to_hex_host(BYTE half_byte_in, char *hex){
    BYTE half_byte_conv = half_byte_in & 0x0f;
    if(half_byte_conv<16){
        if (half_byte_conv>=10){
            *hex = 'a'+ half_byte_conv - 10;
            // printf("%c\n", *hex);
            return;
        }
        else{
            *hex = '0' + half_byte_conv;
            // printf("%c\n", *hex);
            return;
        }
    }
    printf("The half byte must be in range of [0:15]\n");
}
__device__ void word_to_hex_eight(WORD word_in, char *hex_eight, unsigned long hex_eight_size){
    if(hex_eight_size==8){
        half_byte_to_hex(word_in>>28, &hex_eight[0]);
        half_byte_to_hex(word_in>>24, &hex_eight[1]);
        half_byte_to_hex(word_in>>20, &hex_eight[2]);
        half_byte_to_hex(word_in>>16, &hex_eight[3]);
        half_byte_to_hex(word_in>>12, &hex_eight[4]);
        half_byte_to_hex(word_in>>8, &hex_eight[5]);
        half_byte_to_hex(word_in>>4, &hex_eight[6]);
        half_byte_to_hex(word_in, &hex_eight[7]);
        // printf("%c", hex_eight[0]);
        // printf("%d", word_in>>24);
        return;
    }
    printf("The hex_pair must have the length of two characters: %d\n", (int)hex_eight_size);
}

void word_to_hex_eight_host(WORD word_in, char *hex_eight, unsigned long hex_eight_size){
    if(hex_eight_size==8){
        half_byte_to_hex_host(word_in>>28, &hex_eight[0]);
        half_byte_to_hex_host(word_in>>24, &hex_eight[1]);
        half_byte_to_hex_host(word_in>>20, &hex_eight[2]);
        half_byte_to_hex_host(word_in>>16, &hex_eight[3]);
        half_byte_to_hex_host(word_in>>12, &hex_eight[4]);
        half_byte_to_hex_host(word_in>>8, &hex_eight[5]);
        half_byte_to_hex_host(word_in>>4, &hex_eight[6]);
        half_byte_to_hex_host(word_in, &hex_eight[7]);
        // printf("%c", hex_eight[0]);
        // printf("%d", word_in>>24);
        return;
    }
    printf("The hex_pair must have the length of two characters: %d\n", (int)hex_eight_size);
}

__device__ void words_to_hex_string(WORD *words_in, unsigned long words_len, char hex_str[], unsigned long hex_str_len){
    char hex_eight[8];
    if(hex_str_len == 8*words_len){
        for (int i = 0; i<words_len; ++i){
            // printf("\n w: %08x", words_in[i]);
            word_to_hex_eight(words_in[i], hex_eight, sizeof(hex_eight));
            hex_str[8*i] = hex_eight[0];
            hex_str[8*i+1] = hex_eight[1];
            hex_str[8*i+2] = hex_eight[2];
            hex_str[8*i+3] = hex_eight[3];
            hex_str[8*i+4] = hex_eight[4];
            hex_str[8*i+5] = hex_eight[5];
            hex_str[8*i+6] = hex_eight[6];
            hex_str[8*i+7] = hex_eight[7];
            // printf("%c \n", hex_eight[7]);
        }
        // printf("\n%s", hex_str);
        return;
    }
    printf("The hex_string must have the lenght of 4*bytes_len: %d\n", (int)hex_str_len);
}

__device__ void add_two_words_array_512_bit(WORD *a, WORD *b){
    
    for (int i = 15; i>=0; i--){

        a[i] += b[i];

    }
}

__device__ void add_two_words_array_512_bit_with_carry(WORD *a, WORD *b){
    WORD sum = 0;
    WORD sum1 = 0;
    
    for (int i = 15; i>=0; i--){

        sum = ((a[i]&0x0000ffff)+(b[i]&0x0000ffff)+(sum1>>16));
        sum1 = ((a[i]>>16)+(b[i]>>16)+(sum>>16));
        a[i]= (sum & 0x0000ffff) + (sum1<<16);

    }
}

__device__ void print_words_inline(WORD *w, unsigned long w_len){
    printf("\n");
    for (int i = 0; i< w_len; i++){
        printf("%08x", w[i]);
    }
    printf("\n");
}

void print_words_inline_host(WORD *w, unsigned long w_len){
    printf("\n");
    for (int i = 0; i< w_len; i++){
        printf("%08x", w[i]);
    }
    printf("\n");
}

void print_words_multiline_host(WORD *w, unsigned long w_len){
    printf("\n");
    for (int i = 0; i< w_len; i++){
        printf("%08x\n", w[i]);
    }
    printf("\n");
}

__device__ void print_words_multiline(WORD *w, unsigned long w_len){
    printf("\n");
    for (int i = 0; i< w_len; i++){
        printf("%08x\n", w[i]);
    }
    printf("\n");
}

__device__ void endian_cvt(WORD *w){
    WORD out;

    out = (*w>>24)|((*w>>8)&0x0000ff00)|((*w<<8)&0x00ff0000)|(*w<<24);

    *w = out;
}

__device__ void endian_full(WORD *w, unsigned long w_len){
    for (int i = 0; i < w_len; i++)
    {
        endian_cvt(&w[i]);
    }
}
void little_endian(char *c, unsigned long w_len){
    char *dc = (char*)malloc(w_len);

    for (int i = 0; i< w_len; i+=2){
        dc[w_len-2-i] = c[i];
        dc[w_len-1-i] = c[i+1];   
    }
    for (int i = 0; i< w_len; i++){
        c[i] = dc[i];

    }
    c[w_len] = '\0';
    
}

void endian_cvt_host(WORD *w){
    WORD out;

    out = (*w>>24)|((*w>>8)&0x0000ff00)|((*w<<8)&0x00ff0000)|(*w<<24);

    *w = out;
}

 
 // Create init state for SHA-256
 __device__ void sha256_init(SHA256_CTX *ctx)
 {
     ctx->datalen = 0;
     ctx->bitlen = 0;
     ctx->state[0] = h0;
     ctx->state[1] = h1;
     ctx->state[2] = h2;
     ctx->state[3] = h3;
     ctx->state[4] = h4;
     ctx->state[5] = h5;
     ctx->state[6] = h6;
     ctx->state[7] = h7;
 }
 
  // Create init state for SHA-256
  __device__ void sha256_init_words(SHA256_CTX_W *ctx)
  {
      ctx->datalen = 0;
      ctx->bitlen = 0;
      ctx->state[0] = h0;
      ctx->state[1] = h1;
      ctx->state[2] = h2;
      ctx->state[3] = h3;
      ctx->state[4] = h4;
      ctx->state[5] = h5;
      ctx->state[6] = h6;
      ctx->state[7] = h7;
  }
  
 __device__ void sha256_transform(SHA256_CTX *ctx, BYTE data[])
 {
    static const WORD k[64] = {0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5, \
        0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174, \
        0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da, \
        0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967, \
        0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85, \
        0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070, \
        0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3, \
        0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};
 // m is W in hardware design
   WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];
 // Calculate the first 16 m elements.
   for (i = 0, j = 0; i < 16; ++i, j += 4)
     m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);
 // Calculate the remain elements.
   for ( ; i < 64; ++i)
     m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];
 // update the new value of state after each block
   a = ctx->state[0];
   b = ctx->state[1];
   c = ctx->state[2];
   d = ctx->state[3];
   e = ctx->state[4];
   f = ctx->state[5];
   g = ctx->state[6];
   h = ctx->state[7];
 // process 64 rounds
   for (i = 0; i < 64; ++i) {
     t1 = h + EP1(e) + CH(e,f,g) + k[i] + m[i];
     t2 = EP0(a) + MAJ(a,b,c);
     h = g;
     g = f;
     f = e;
     e = d + t1;
     d = c;
     c = b;
     b = a;
     a = t1 + t2;
   }
 
   ctx->state[0] += a;
   ctx->state[1] += b;
   ctx->state[2] += c;
   ctx->state[3] += d;
   ctx->state[4] += e;
   ctx->state[5] += f;
   ctx->state[6] += g;
   ctx->state[7] += h;
 }

 __device__ void sha256_transform_words(SHA256_CTX_W *ctx, WORD data[])
 {
    static const WORD k[64] = {0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5, \
        0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174, \
        0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da, \
        0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967, \
        0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85, \
        0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070, \
        0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3, \
        0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};
 // m is W in hardware design
   WORD a, b, c, d, e, f, g, h, i, t1, t2, m[64];
 // Calculate the first 16 m elements.
   for (i = 0; i < 16; ++i)
     m[i] = data[i];
 // Calculate the remain elements.
   for ( ; i < 64; ++i)
     m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];
 // update the new value of state after each block
   a = ctx->state[0];
   b = ctx->state[1];
   c = ctx->state[2];
   d = ctx->state[3];
   e = ctx->state[4];
   f = ctx->state[5];
   g = ctx->state[6];
   h = ctx->state[7];
 // process 64 rounds
   for (i = 0; i < 64; ++i) {
     t1 = h + EP1(e) + CH(e,f,g) + k[i] + m[i];
     t2 = EP0(a) + MAJ(a,b,c);
     h = g;
     g = f;
     f = e;
     e = d + t1;
     d = c;
     c = b;
     b = a;
     a = t1 + t2;
   }
 
   ctx->state[0] += a;
   ctx->state[1] += b;
   ctx->state[2] += c;
   ctx->state[3] += d;
   ctx->state[4] += e;
   ctx->state[5] += f;
   ctx->state[6] += g;
   ctx->state[7] += h;
 }
 // the total length of the message has to be specified
 __device__ void sha256_update(SHA256_CTX *ctx,  BYTE data[], size_t len)
 {
     WORD i;
 
     for (i = 0; i < len; ++i){
         ctx->data[ctx->datalen] = data[i];      // Pad data (message) for each 512-block in --> transform
         ctx->datalen++;
         // after browse for 64 bytes (512-bit block) -> transform the block.
         if(ctx->datalen == 64){
             sha256_transform(ctx, ctx->data);
             ctx->bitlen += 512; // increase the bit length by 512
             ctx->datalen = 0;
         }
     }
 }

 __device__ void sha256_update_words(SHA256_CTX_W *ctx, WORD data[], size_t len)
 {
     WORD i;
 
     for (i = 0; i < len; ++i){
         ctx->data[ctx->datalen] = data[i];      // Pad data (message) for each 512-block in --> transform
         ctx->datalen++;
         // after browse for 64 bytes (512-bit block) -> transform the block.
         if(ctx->datalen == 16){
             sha256_transform_words(ctx, ctx->data);
             ctx->bitlen += 512; // increase the bit length by 512
             ctx->datalen = 0;
         }
     }
 }
 // this function processes for the last block -> after all real data is browsed 
 __device__ void sha256_final(SHA256_CTX *ctx, WORD *hash){
     WORD i;
 // padding is processed from here
     i = ctx->datalen;
     if (ctx->datalen < 56){
         // add byte 0x80 at the first if the datalength is lower than 56
         ctx->data[i++] = 0x80;
         // pad the zero bytes until the byte 56th
         while (i<56)
         {
             ctx->data[i++]=0x00;
         }
     }
     else{
         // add byte at the first
         ctx->data[i++]=0x80;
         // pad zero bytes until the last block
         while (i<64){
             ctx->data[i++]=0x00;
         }
         // transform this block --> it's not the last block
         sha256_transform(ctx, ctx->data);
         // set 56 zero bytes from last_block[0:55]
         memset(ctx->data, 0, 56);
     }
 
     // Append to the padding the total message's length in bits and transform.
     ctx->bitlen += ctx->datalen * 8;
     ctx->data[63] = ctx->bitlen;
     ctx->data[62] = ctx->bitlen >> 8;
     ctx->data[61] = ctx->bitlen >> 16;
     ctx->data[60] = ctx->bitlen >> 24;
     ctx->data[59] = ctx->bitlen >> 32;
     ctx->data[58] = ctx->bitlen >> 40;
     ctx->data[57] = ctx->bitlen >> 48;
     ctx->data[56] = ctx->bitlen >> 56;
 // end padding
     sha256_transform(ctx, ctx->data);
 
     // Since this implementation uses little endian byte ordering and SHA uses big endian,
     // reverse all the bytes when copying the final state to the output hash.
 
 
         hash[0] = ctx->state[0];
         hash[1] = ctx->state[1];
         hash[2] = ctx->state[2];
         hash[3] = ctx->state[3];
         hash[4] = ctx->state[4];
         hash[5] = ctx->state[5];
         hash[6] = ctx->state[6];
         hash[7] = ctx->state[7]; 
  
 }
 
  // this function processes for the last block -> after all real data is browsed 
  __device__ void sha256_final_words(SHA256_CTX_W *ctx, WORD *hash){
    WORD i;
// padding is processed from here
    i = ctx->datalen;
    if (ctx->datalen < 14){
        // add byte 0x80 at the first if the datalength is lower than 56
        ctx->data[i++] = 0x80000000;
        // pad the zero bytes until the byte 56th
        while (i<14)
        {
            ctx->data[i++]=0x00000000;
        }
    }
    else{
        // add bit 1 at the first
        ctx->data[i++]=0x80000000;
        // pad zero bytes until the last block
        while (i<16){
            ctx->data[i++]=0x00000000;
        }
        // transform this block --> it's not the last block
        sha256_transform_words(ctx, ctx->data);
        // set 56 zero bytes from last_block[0:55]
        ctx->data[0] =0x00000000;
        ctx->data[1] =0x00000000;
        ctx->data[2] =0x00000000;
        ctx->data[3] =0x00000000;
        ctx->data[4] =0x00000000;
        ctx->data[5] =0x00000000;
        ctx->data[6] =0x00000000;
        ctx->data[7] =0x00000000;
        ctx->data[8] =0x00000000;
        ctx->data[9] =0x00000000;
        ctx->data[10]=0x00000000;
        ctx->data[11]=0x00000000;
        ctx->data[12]=0x00000000;
        ctx->data[13]=0x00000000;
    }

    // Append to the padding the total message's length in bits and transform.
    ctx->bitlen += ctx->datalen * 32;
    ctx->data[15] = ctx->bitlen;
    ctx->data[14] = ctx->bitlen>>32;


// end padding
    sha256_transform_words(ctx, ctx->data);

    // Since this implementation uses little endian byte ordering and SHA uses big endian,
    // reverse all the bytes when copying the final state to the output hash.


        hash[0] = ctx->state[0];
        hash[1] = ctx->state[1];
        hash[2] = ctx->state[2];
        hash[3] = ctx->state[3];
        hash[4] = ctx->state[4];
        hash[5] = ctx->state[5];
        hash[6] = ctx->state[6];
        hash[7] = ctx->state[7]; 
 
}

 __device__ void sha256_in_words(SHA256_CTX_W *ctx, WORD *words_in, unsigned long words_in_len, WORD *hash_w){

    // printf("\n%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", words_in[0], words_in[1], words_in[2], words_in[3], words_in[4], words_in[5], words_in[6], words_in[7], words_in[8], words_in[9], words_in[10], words_in[11], words_in[12], words_in[13], words_in[14], words_in[15], words_in[16], words_in[17], words_in[18], words_in[19], words_in[20], words_in[21], words_in[22], words_in[23]); // true

     sha256_init_words(ctx);
     
     sha256_update_words(ctx, words_in, words_in_len);
    //  print_words_inline(ctx->state, 8);
     sha256_final_words(ctx, hash_w);
     
 }
 
 __device__ void sha256_in_words_org(SHA256_CTX *ctx, WORD *words_in, unsigned long words_in_len, WORD *hash_w){
     unsigned long bytes_in_len = words_in_len * 4;
    //  printf("%d\n", words_in_len);
    // printf("\n%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", words_in[0], words_in[1], words_in[2], words_in[3], words_in[4], words_in[5], words_in[6], words_in[7], words_in[8], words_in[9], words_in[10], words_in[11], words_in[12], words_in[13], words_in[14], words_in[15], words_in[16], words_in[17], words_in[18], words_in[19], words_in[20], words_in[21], words_in[22], words_in[23]); // true
    BYTE *bytes_in = (BYTE *)malloc((bytes_in_len)*sizeof(BYTE));
     for (int i = 0; i<words_in_len; i++){
         bytes_in[4*i] = words_in[i] >> 24;
         bytes_in[4*i+1] = words_in[i] >> 16;
         bytes_in[4*i+2] = words_in[i] >> 8;
         bytes_in[4*i+3] = words_in[i];
        //  printf("%x %x %x %x ", bytes_in[4*i], bytes_in[4*i+1], bytes_in[4*i+2], bytes_in[4*i+3]);
     }
     sha256_init(ctx);
     
     sha256_update(ctx, bytes_in, bytes_in_len);
    //  print_words_inline(ctx->state, 8);
     sha256_final(ctx, hash_w);
     
 }
 __device__ void sha256_in_words_test(SHA256_CTX_W *ctx, WORD *words_in, unsigned long words_in_len, WORD *hash_w){

   //  printf("%d\n", words_in_len);
   // printf("\n%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", words_in[0], words_in[1], words_in[2], words_in[3], words_in[4], words_in[5], words_in[6], words_in[7], words_in[8], words_in[9], words_in[10], words_in[11], words_in[12], words_in[13], words_in[14], words_in[15], words_in[16], words_in[17], words_in[18], words_in[19], words_in[20], words_in[21], words_in[22], words_in[23]); // true
   
   

    sha256_init_words(ctx);
    
    sha256_update_words(ctx, words_in, words_in_len);
   //  print_words_inline(ctx->state, 8);
    sha256_final_words(ctx, hash_w);
    
}


__device__ void hmac(SHA256_CTX_W *ctx, WORD *salt, unsigned long salt_len, WORD *message, unsigned long message_len, WORD* out_hmac){
     WORD IPAD[8] = {ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm}; // 256-bit 363636...36   
     WORD OPAD[8] = {opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm}; // 256-bit 5c5c5c...5c
    WORD *khash = (WORD*) malloc(sizeof(WORD)*8);
    // print_words_inline(message, 20); // OK
 
    sha256_in_words(ctx, message, message_len, khash);
    
    // print_words_inline(khash, 8); // OK
    // for(int i=0;i<8; i++){
    //     printf("%08x", khash[i]);
    // }
    WORD ixor[16] = {\
        IPAD[0]^khash[0],\
        IPAD[1]^khash[1],\
        IPAD[2]^khash[2],\
        IPAD[3]^khash[3],\
        IPAD[4]^khash[4],\
        IPAD[5]^khash[5],\
        IPAD[6]^khash[6],\
        IPAD[7]^khash[7],\
        IPAD[0],\
        IPAD[1],\
        IPAD[2],\
        IPAD[3],\
        IPAD[4],\
        IPAD[5],\
        IPAD[6],\
        IPAD[7],\
    };

    WORD oxor[16] = {\
        OPAD[0]^khash[0],\
        OPAD[1]^khash[1],\
        OPAD[2]^khash[2],\
        OPAD[3]^khash[3],\
        OPAD[4]^khash[4],\
        OPAD[5]^khash[5],\
        OPAD[6]^khash[6],\
        OPAD[7]^khash[7],\
        OPAD[0],\
        OPAD[1],\
        OPAD[2],\
        OPAD[3],\
        OPAD[4],\
        OPAD[5],\
        OPAD[6],\
        OPAD[7],\
    };
    WORD *in_ihash = (WORD*)malloc((sizeof(ixor)/sizeof(WORD)+salt_len)*sizeof(WORD));
    unsigned long in_ihash_len = sizeof(ixor)/sizeof(WORD)+salt_len;

    int i;
    for(i = 0; i<sizeof(ixor)/sizeof(WORD); i++){
        in_ihash[i] = ixor[i];
    }
   
    for(;i<sizeof(ixor)/sizeof(WORD)+salt_len; i++){
        in_ihash[i] = salt[i-sizeof(ixor)/sizeof(WORD)];
    }
    
    WORD *ihash = (WORD*)malloc(8*sizeof(WORD));
    // print_words_inline(in_ihash, sizeof(ixor)/sizeof(WORD)+salt_len); // Problem
    sha256_in_words(ctx, in_ihash, in_ihash_len, ihash); // Why it's wrong
    // print_words_inline(ihash, 8); // OK
    unsigned long in_ohash_len = sizeof(oxor)/sizeof(WORD)+8;
    WORD *in_ohash = (WORD*)malloc(in_ohash_len*sizeof(WORD));  // WORD[24]

    // printf("%d\n", (in_ohash_len));
    // printf("%d\n", in_ohash_len); // 24 --> true
    for(i = 0; i<sizeof(oxor)/sizeof(WORD); i++){
        in_ohash[i] = oxor[i];
    }
    for(;i<sizeof(ixor)/sizeof(WORD)+salt_len; i++){
        in_ohash[i] = ihash[i-sizeof(oxor)/sizeof(WORD)];
    }

    // printf("%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x %d\n", in_ohash[0], in_ohash[1], in_ohash[2], in_ohash[3], in_ohash[4], in_ohash[5], in_ohash[6], in_ohash[7], in_ohash[8], in_ohash[9], in_ohash[10], in_ohash[11], in_ohash[12], in_ohash[13], in_ohash[14], in_ohash[15], in_ohash[16], in_ohash[17], in_ohash[18], in_ohash[19], in_ohash[20], in_ohash[21], in_ohash[22], in_ohash[23], in_ohash_len); // true

    WORD temp[] = {in_ohash[0], in_ohash[1], in_ohash[2], in_ohash[3], in_ohash[4], in_ohash[5], in_ohash[6], in_ohash[7], in_ohash[8], in_ohash[9], in_ohash[10], in_ohash[11], in_ohash[12], in_ohash[13], in_ohash[14], in_ohash[15], in_ohash[16], in_ohash[17], in_ohash[18], in_ohash[19], in_ohash[20], in_ohash[21], in_ohash[22], in_ohash[23]};
    // static WORD ohash[8];
    // printf("\n");
    sha256_in_words(ctx, temp, in_ohash_len, out_hmac);
    // print_words_inline(ohash, 8); // OK
    
    // return ohash;
}
__device__ void hmac_2(SHA256_CTX_W *ctx, WORD *salt, unsigned long salt_len, WORD *message, unsigned long message_len, WORD* out_hmac, WORD khash[8], WORD in_ihash[49], WORD ihash[8], WORD in_ohash[24]){
     WORD IPAD[8] = {ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm}; // 256-bit 363636...36   
     WORD OPAD[8] = {opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm}; // 256-bit 5c5c5c...5c

    // print_words_inline(message, 20); // OK
 
    // sha256_in_words(ctx, message, message_len, khash);
    
    // print_words_inline(khash, 8); // OK
    // for(int i=0;i<8; i++){
    //     printf("%08x", khash[i]);
    // }
    WORD ixor[16] = {\
        IPAD[0]^khash[0],\
        IPAD[1]^khash[1],\
        IPAD[2]^khash[2],\
        IPAD[3]^khash[3],\
        IPAD[4]^khash[4],\
        IPAD[5]^khash[5],\
        IPAD[6]^khash[6],\
        IPAD[7]^khash[7],\
        IPAD[0],\
        IPAD[1],\
        IPAD[2],\
        IPAD[3],\
        IPAD[4],\
        IPAD[5],\
        IPAD[6],\
        IPAD[7],\
    };

    WORD oxor[16] = {\
        OPAD[0]^khash[0],\
        OPAD[1]^khash[1],\
        OPAD[2]^khash[2],\
        OPAD[3]^khash[3],\
        OPAD[4]^khash[4],\
        OPAD[5]^khash[5],\
        OPAD[6]^khash[6],\
        OPAD[7]^khash[7],\
        OPAD[0],\
        OPAD[1],\
        OPAD[2],\
        OPAD[3],\
        OPAD[4],\
        OPAD[5],\
        OPAD[6],\
        OPAD[7],\
    };

    unsigned long in_ihash_len = sizeof(ixor)/sizeof(WORD)+salt_len;
    // printf("%d\n", (sizeof(ixor)/sizeof(WORD)+salt_len));
    int i;
    for(i = 0; i<sizeof(ixor)/sizeof(WORD); i++){
        in_ihash[i] = ixor[i];
    }
   
    for(;i<sizeof(ixor)/sizeof(WORD)+salt_len; i++){
        in_ihash[i] = salt[i-sizeof(ixor)/sizeof(WORD)];
    }
    

    // print_words_inline(in_ihash, sizeof(ixor)/sizeof(WORD)+salt_len); // Problem
    sha256_in_words(ctx, in_ihash, in_ihash_len, ihash); // Why it's wrong
    // print_words_inline(ihash, 8); // OK
    unsigned long in_ohash_len = sizeof(oxor)/sizeof(WORD)+8;


    
    // printf("%d\n", in_ohash_len); // 24 --> true
    for(i = 0; i<sizeof(oxor)/sizeof(WORD); i++){
        in_ohash[i] = oxor[i];
    }
    for(;i<sizeof(ixor)/sizeof(WORD)+salt_len; i++){
        in_ohash[i] = ihash[i-sizeof(oxor)/sizeof(WORD)];
    }

    // printf("%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x %d\n", in_ohash[0], in_ohash[1], in_ohash[2], in_ohash[3], in_ohash[4], in_ohash[5], in_ohash[6], in_ohash[7], in_ohash[8], in_ohash[9], in_ohash[10], in_ohash[11], in_ohash[12], in_ohash[13], in_ohash[14], in_ohash[15], in_ohash[16], in_ohash[17], in_ohash[18], in_ohash[19], in_ohash[20], in_ohash[21], in_ohash[22], in_ohash[23], in_ohash_len); // true

    WORD temp[] = {in_ohash[0], in_ohash[1], in_ohash[2], in_ohash[3], in_ohash[4], in_ohash[5], in_ohash[6], in_ohash[7], in_ohash[8], in_ohash[9], in_ohash[10], in_ohash[11], in_ohash[12], in_ohash[13], in_ohash[14], in_ohash[15], in_ohash[16], in_ohash[17], in_ohash[18], in_ohash[19], in_ohash[20], in_ohash[21], in_ohash[22], in_ohash[23]};
    // static WORD ohash[8];
    // printf("\n");
    sha256_in_words(ctx, temp, in_ohash_len, out_hmac);
    // print_words_inline(ohash, 8); // OK
    
    // return ohash;
}

__device__ void hmac_test(SHA256_CTX_W *ctx, WORD *salt, unsigned long salt_len, WORD *message, unsigned long message_len, WORD* out_hmac, WORD khash[8], WORD in_ihash[37], WORD ihash[8], WORD in_ohash[24]){
    // WORD IPAD[8] = {ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm}; // 256-bit 363636...36   
    // WORD OPAD[8] = {opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm}; // 256-bit 5c5c5c...5c

    // print_words_inline(message, 20); // OK
    
    sha256_in_words(ctx, message, message_len, khash);
    
    // print_words_inline(khash, 8); // OK
    // for(int i=0;i<8; i++){
    //     printf("%08x", khash[i]);
    // }
    
    WORD ixor[16] = {\
        0x36363636^khash[0],\
        0x36363636^khash[1],\
        0x36363636^khash[2],\
        0x36363636^khash[3],\
        0x36363636^khash[4],\
        0x36363636^khash[5],\
        0x36363636^khash[6],\
        0x36363636^khash[7],\
        0x36363636,\
        0x36363636,\
        0x36363636,\
        0x36363636,\
        0x36363636,\
        0x36363636,\
        0x36363636,\
        0x36363636,\
    };

    WORD oxor[16] = {\
        0x5C5C5C5C^khash[0],\
        0x5C5C5C5C^khash[1],\
        0x5C5C5C5C^khash[2],\
        0x5C5C5C5C^khash[3],\
        0x5C5C5C5C^khash[4],\
        0x5C5C5C5C^khash[5],\
        0x5C5C5C5C^khash[6],\
        0x5C5C5C5C^khash[7],\
        0x5C5C5C5C,\
        0x5C5C5C5C,\
        0x5C5C5C5C,\
        0x5C5C5C5C,\
        0x5C5C5C5C,\
        0x5C5C5C5C,\
        0x5C5C5C5C,\
        0x5C5C5C5C,\
    };
    // print_words_inline(IPAD, 8);
    // print_words_inline(OPAD, 8);
    unsigned long in_ihash_len = 37;
    int i;
    
    for(i = 0; i<sizeof(ixor)/sizeof(WORD); i++){
        in_ihash[i] = ixor[i];
    }
   
    for(;i<sizeof(ixor)/sizeof(WORD)+salt_len; i++){
        in_ihash[i] = salt[i-sizeof(ixor)/sizeof(WORD)];
    }
    
    // print_words_inline(in_ihash, sizeof(ixor)/sizeof(WORD)+salt_len); // Problem
    
    sha256_in_words(ctx, in_ihash, in_ihash_len, ihash); // Why it's wrong
    // printf("OK hmac_tes before in_ohash_len\n");
    // print_words_inline(ihash, 8); // OK
    unsigned long in_ohash_len = 24;
 

    
    // printf("in_ohash: %u\n", in_ohash); // 24 --> true
    for(i = 0; i<sizeof(oxor)/sizeof(WORD); i++){
        in_ohash[i] = oxor[i];
        // in_ohash[i] = 0x12341234;
    }
    for(;i<sizeof(ixor)/sizeof(WORD)+salt_len; i++){
        in_ohash[i] = ihash[i-sizeof(oxor)/sizeof(WORD)];
    }
    // printf("OK hmac_test in_ohash_len\n");
    // printf("%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x %d\n", in_ohash[0], in_ohash[1], in_ohash[2], in_ohash[3], in_ohash[4], in_ohash[5], in_ohash[6], in_ohash[7], in_ohash[8], in_ohash[9], in_ohash[10], in_ohash[11], in_ohash[12], in_ohash[13], in_ohash[14], in_ohash[15], in_ohash[16], in_ohash[17], in_ohash[18], in_ohash[19], in_ohash[20], in_ohash[21], in_ohash[22], in_ohash[23], in_ohash_len); // true

    WORD temp[] = {in_ohash[0], in_ohash[1], in_ohash[2], in_ohash[3], in_ohash[4], in_ohash[5], in_ohash[6], in_ohash[7], in_ohash[8], in_ohash[9], in_ohash[10], in_ohash[11], in_ohash[12], in_ohash[13], in_ohash[14], in_ohash[15], in_ohash[16], in_ohash[17], in_ohash[18], in_ohash[19], in_ohash[20], in_ohash[21], in_ohash[22], in_ohash[23]};
    // static WORD ohash[8];
    // printf("\n");
    
    sha256_in_words(ctx, temp, in_ohash_len, out_hmac);
    
    // print_words_inline(out_hmac, 8); // OK
    
    // return ohash;
}
__device__ void pbkdf2(SHA256_CTX_W *ctx, WORD *block, unsigned long block_len, int dklenP, WORD *pbkdf2_out, WORD salt[21]){
    int num_loop = 1024/dklenP;
    // WORD *salt = (WORD*)malloc((block_len+1)*sizeof(WORD));
    WORD *hmac_out = (WORD*)malloc(8*sizeof(WORD));
    // int hmac_out_len = 8;
    
    // printf("pbkdf2: %08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", block[0], block[1], block[2], block[3], block[4], block[5], block[6], block[7], block[8], block[9], block[10], block[11], block[12], block[13], block[14], block[15], block[16], block[17], block[18], block[19]);
    for(int i = 0; i<block_len; i++){
        salt[i]=block[i];
    }
    
    for (int i = 1; i <= num_loop; i++)
    {
        salt[block_len] = i;
        hmac(ctx, salt, block_len+1, block, block_len, hmac_out);
        // print_words_inline(hmac_out, 8); // False 
        for(int j = 0; j<8; j++){
            pbkdf2_out[(i-1)*8+j] = hmac_out[j];
        }
        // printf("%08x%08x%08x%08x%08x%08x%08x%08x\n", hmac_out[0], hmac_out[1], hmac_out[2], hmac_out[3], hmac_out[4], hmac_out[5], hmac_out[6], hmac_out[7]); // wrong
        // }
    } 
}
__device__ void pbkdf2_test(SHA256_CTX_W *ctx, WORD block[20], unsigned long block_len, WORD pbkdf2_out[32], WORD salt[21], WORD hmac_out[8], WORD khash[8], WORD in_ihash[37], WORD ihash[8], WORD in_ohash[24]){

    // printf("OK\n");

    // int hmac_out_len = 8;
    
    
    // printf("pbkdf2: %08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", block[0], block[1], block[2], block[3], block[4], block[5], block[6], block[7], block[8], block[9], block[10], block[11], block[12], block[13], block[14], block[15], block[16], block[17], block[18], block[19]);
    for(int i = 0; i<block_len; i++){
        salt[i]=block[i];
    }
    // printf("pbkdf2_out: %u\n", pbkdf2_out); // 24 --> true
    // print_words_multiline(pbkdf2_out, 32);
    for (int i = 1; i <= 4; i++)
    {
        salt[block_len] = i;
        // printf("OK SALT\n");
        hmac_test(ctx, salt, block_len+1, block, block_len, hmac_out, khash, in_ihash, ihash, in_ohash);
        // printf("OK HMAC\n");
        // print_words_inline(hmac_out, 8); // False 
        for(int j = 0; j<8; j++){
            pbkdf2_out[(i-1)*8+j] = hmac_out[j];
            // pbkdf2_out[(i-1)*8+j] = 0xffffffff;
            // printf("%08x, %d\n", pbkdf2_out[(i-1)*8+j], (i-1)*8+j);
            // print_words_multiline(pbkdf2_out, 32);
        }
        // printf("%08x%08x%08x%08x%08x%08x%08x%08x\n", hmac_out[0], hmac_out[1], hmac_out[2], hmac_out[3], hmac_out[4], hmac_out[5], hmac_out[6], hmac_out[7]); // wrong
        // }
        
    
    } 
    // print_words_multiline(pbkdf2_out, 32);

}

__device__ void pbkdf2_2nd(SHA256_CTX_W *ctx, WORD *rm_out, unsigned long rm_out_len, WORD *block, unsigned long block_len, WORD* pbkdf2_out, WORD salt[21], WORD hmac_out[8], WORD khash[8], WORD in_ihash[37], WORD ihash[8], WORD in_ohash[24]){
  

    // int hmac_out_len = 8;
    
    for(int i = 0; i<rm_out_len; i++){
        salt[i]=rm_out[i];
    }


    salt[rm_out_len] = 1;
    
    hmac_2(ctx, salt, rm_out_len+1, block, block_len, hmac_out, khash, in_ihash, ihash, in_ohash);
    
    pbkdf2_out[0] = hmac_out[0];
    for(int j = 0; j<8; j++){
        //pbkdf2_out[(i-1)*8+j] = hmac_out[j];
        pbkdf2_out[j] = hmac_out[j];
        //printf("%d: %08x \n", j, hmac_out[j]);
        // printf("%d: %08x \n", j, hmac_out[j]);
    }

}
__device__ void salsa_mix_func(WORD *des, WORD *a1, WORD *a2, WORD b){
    // printf("%08x \n", *a1);
    WORD sum = *a1 + *a2;
    // printf("0x%08x + 0x%08x = 0x%08x \n", *a1, *a2,  sum);
    WORD rotl = (sum<<b) | (sum>>(32-b));
    WORD xorv = *des ^ rotl;
    
    *des = xorv;
    
}
__device__ void salsa_round(WORD *x1, WORD *x2, WORD *x3, WORD *x4){
    salsa_mix_func(x1, x4, x3, 7);
    // printf("%08x \n", *x1);
    salsa_mix_func(x2, x1, x4, 9);
    salsa_mix_func(x3, x2, x1, 13);
    salsa_mix_func(x4, x3, x2, 18);
}

__device__ WORD * salsa20_8(WORD *x){
    static WORD out[16];
    // for(int i = 0; i<4; i++){
    for(int i = 0; i<4; i++){
        // if(i==0){
        //     printf("%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", x[0], x[1], x[2], x[3], x[4], x[5], x[6], x[7], x[8], x[9], x[10], x[11], x[12], x[13], x[14], x[15] );
        // }
        salsa_round(&x[4], &x[8], &x[12], &x[0]);
        salsa_round(&x[9], &x[13], &x[1], &x[5]);
        salsa_round(&x[14], &x[2], &x[6], &x[10]);
        salsa_round(&x[3], &x[7], &x[11], &x[15]);
        salsa_round(&x[1], &x[2], &x[3], &x[0]);
        salsa_round(&x[6], &x[7], &x[4], &x[5]);
        salsa_round(&x[11], &x[8], &x[9], &x[10]);
        salsa_round(&x[12], &x[13], &x[14], &x[15]);
    
        
    }
    for(int i=0; i<16; i++){
        out[i] = x[i];
    }
    return out;
}

__device__ void  blockmix(WORD *block){
    WORD x_arr[16];
    WORD x_arr_cpy[16];
    // printf("%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", block[0], block[1], block[2], block[3], block[4], block[5], block[6], block[7], block[8], block[9], block[10], block[11], block[12], block[13], block[14], block[15]);
    for (int i = 0; i < 16; i++){
        x_arr[i] = block[i];                                        // 1
 
    }
    // printf("%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", x_arr[0], x_arr[1], x_arr[2], x_arr[3], x_arr[4], x_arr[5], x_arr[6], x_arr[7], x_arr[8], x_arr[9], x_arr[10], x_arr[11], x_arr[12], x_arr[13], x_arr[14], x_arr[15]);
    for (int i = 0; i<2; i++){
        for (int j = 0; j < 16; j++){
            x_arr_cpy[j] = x_arr[j] ^ block[j+16];                  // 2


            x_arr[j] ^= block[j+16];                                // 3

        }
        // printf("xcp %08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", x_arr_cpy[0], x_arr_cpy[1], x_arr_cpy[2], x_arr_cpy[3], x_arr_cpy[4], x_arr_cpy[5], x_arr_cpy[6], x_arr_cpy[7], x_arr_cpy[8], x_arr_cpy[9], x_arr_cpy[10], x_arr_cpy[11], x_arr_cpy[12], x_arr_cpy[13], x_arr_cpy[14], x_arr_cpy[15] );
        salsa20_8(x_arr_cpy);
  
        add_two_words_array_512_bit(x_arr, x_arr_cpy);   // 4
        // printf("0x%08x + 0x%08x = 0x%08x\n", a[0], x_arr_cpy[0], x_arr[0]);
        for (int j = 0; j < 16; j++){

            block[(16*i)+j] = x_arr[j];                             // 5

        }
    }
}

__device__ void romix(WORD *block, int N, WORD mem[1024][32]){
    // WORD mem[1024][32];
    
    int j;
     for (int i = 0; i<N; i++){
        for (j = 0; j < 32; j++){
            mem[i][j] = block[j];
            //printf("N: %d, j: %d, i: %d, \n",N, j ,i);
        }
        // if(i == 1023){
            // printf("i: %d, \n",N, i);
        // }
        
        blockmix(block);
        
    }
    
    // printf("%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", block[0], block[1], block[2], block[3], block[4], block[5], block[6], block[7], block[8], block[9], block[10], block[11], block[12], block[13], block[14], block[15], block[16], block[17], block[18], block[19], block[20], block[21], block[22], block[23], block[24], block[25], block[26], block[27], block[28], block[29], block[30], block[31]);
    for (int i = 0; i<N; i++){
        j = (block[16] & 0x000003ff);
 
        for (int k = 0; k<32; k++){
            // int a = block[k] ^ mem[j][k];
            // printf("j: %u, i: %u, k: %u\n",j ,i, k);
            block[k] ^= mem[j][k];
            // if(a != block[k])
            
        }
        blockmix(block);
    }
    

}

__device__ void scrypt( SHA256_CTX_W *ctx,\
                        WORD *block, \
                        unsigned long block_len, \
                        WORD scrypt_out[8], \
                        WORD mem[1024][32], \
                        WORD salt[21], \
                        WORD hmac_out[8], \
                        WORD khash[8], \
                        WORD in_ihash[37], \
                        WORD in_2_ihash[49], \
                        WORD ihash[8], \
                        WORD in_ohash[24], \
                        WORD pbkdf2_rm_out[32]){

    // WORD *pbkdf2_1_out = (WORD*)malloc(pbkdf2_out_len_1*sizeof(WORD));
    // printf("%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", block[0], block[1], block[2], block[3], block[4], block[5], block[6], block[7], block[8], block[9], block[10], block[11], block[12], block[13], block[14], block[15], block[16], block[17], block[18], block[19]);
    
    // print_words_inline(&block[19], 1);
    // print_words_inline(block, 20); // OK
    pbkdf2_test(ctx, block, block_len, pbkdf2_rm_out, salt, hmac_out, khash, in_ihash, ihash, in_ohash);
    // printf("%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", pbkdf2_rm_out[0], pbkdf2_rm_out[1], pbkdf2_rm_out[2], pbkdf2_rm_out[3], pbkdf2_rm_out[4], pbkdf2_rm_out[5], pbkdf2_rm_out[6], pbkdf2_rm_out[7], pbkdf2_rm_out[8], pbkdf2_rm_out[9], pbkdf2_rm_out[10], pbkdf2_rm_out[11], pbkdf2_rm_out[12], pbkdf2_rm_out[13], pbkdf2_rm_out[14], pbkdf2_rm_out[15], pbkdf2_rm_out[16], pbkdf2_rm_out[17], pbkdf2_rm_out[18], pbkdf2_rm_out[19], pbkdf2_rm_out[20], pbkdf2_rm_out[21], pbkdf2_rm_out[22], pbkdf2_rm_out[23], pbkdf2_rm_out[24], pbkdf2_rm_out[25], pbkdf2_rm_out[26], pbkdf2_rm_out[27], pbkdf2_rm_out[28], pbkdf2_rm_out[29], pbkdf2_rm_out[30], pbkdf2_rm_out[31]);
    // printf("OK\n");
    // print_words_multiline(pbkdf2_rm_out, 32);
    endian_full(pbkdf2_rm_out, 32);
  
    
    romix(pbkdf2_rm_out, 1024, mem);
    
    
    // printf("%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", pbkdf2_rm_out[0], pbkdf2_rm_out[1], pbkdf2_rm_out[2], pbkdf2_rm_out[3], pbkdf2_rm_out[4], pbkdf2_rm_out[5], pbkdf2_rm_out[6], pbkdf2_rm_out[7], pbkdf2_rm_out[8], pbkdf2_rm_out[9], pbkdf2_rm_out[10], pbkdf2_rm_out[11], pbkdf2_rm_out[12], pbkdf2_rm_out[13], pbkdf2_rm_out[14], pbkdf2_rm_out[15], pbkdf2_rm_out[16], pbkdf2_rm_out[17], pbkdf2_rm_out[18], pbkdf2_rm_out[19], pbkdf2_rm_out[20], pbkdf2_rm_out[21], pbkdf2_rm_out[22], pbkdf2_rm_out[23], pbkdf2_rm_out[24], pbkdf2_rm_out[25], pbkdf2_rm_out[26], pbkdf2_rm_out[27], pbkdf2_rm_out[28], pbkdf2_rm_out[29], pbkdf2_rm_out[30], pbkdf2_rm_out[31]);
    
    endian_full(pbkdf2_rm_out, 32);
    // print_words_multiline(pbkdf2_1_out, 32);
    
    pbkdf2_2nd(ctx, pbkdf2_rm_out, 32, block, block_len, scrypt_out, salt, hmac_out, khash, in_2_ihash, ihash, in_ohash);

    // printf("%08x%08x%08x%08x%08x%08x%08x%08x\n", pbkdf2_2_out[0],pbkdf2_2_out[1],pbkdf2_2_out[2],pbkdf2_2_out[3],pbkdf2_2_out[4],pbkdf2_2_out[5],pbkdf2_2_out[6],pbkdf2_2_out[7]);
    // print_words_inline(pbkdf2_2_out, 20);
}






hipError_t scryptWithCuda(SCRYPT_PKG **pkgs, int block_per_grid, int thread_per_block);




__global__ void scryptCuda(SCRYPT_PKG **pkgs)
{   
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;
    // int blockId = blockIdx.y*gridDim.x+blockIdx.x;
    // uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    // uint32_t stride = blockDim.x * gridDim.x;
    // char hex_str[65];
    SHA256_CTX *ctx = new SHA256_CTX();
    // int i;
    
    // for (i = 0; i<3000000000; i++){
       
    // }
    //WORD T_Nonce = pkgs[threadId]->input[19];
    
    if(threadId < NUMTHREAD){
        // for (uint32_t threadId = index; threadId < MAXNONCE; threadId += stride){
            WORD T_Nonce = pkgs[threadId]->input[19];
            endian_cvt(&T_Nonce);
            // if(threadId == 1024)
                // printf("Thread id %d: %08x maxnonce: %u\n", threadId, T_Nonce, MAXNONCE/NUMTHREAD);
        for(int i = 0; i<MAXNONCE/NUMTHREAD; i++){
            // if(threadId == NUMTHREAD-1 && i==MAXNONCE/NUMTHREAD-1){
            //     // print_words_inline(&in[20*threadId+19], 1);
            //     // printf("Thread ID (%d) = %08x%08x%08x%08x%08x%08x%08x%08x\n", threadId, out[8*threadId],out[8*threadId+1],out[8*threadId+2],out[8*threadId+3],out[8*threadId+4],out[8*threadId+5],out[8*threadId+6],out[8*threadId+7]);
            // }
            // printf("%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x%08x\n", pkgs[threadId]->input[0],pkgs[threadId]->input[1],pkgs[threadId]->input[2],pkgs[threadId]->input[3],pkgs[threadId]->input[4],pkgs[threadId]->input[5],pkgs[threadId]->input[6],pkgs[threadId]->input[7], pkgs[threadId]->input[8], pkgs[threadId]->input[9], pkgs[threadId]->input[10], pkgs[threadId]->input[11], pkgs[threadId]->input[12], pkgs[threadId]->input[13], pkgs[threadId]->input[14], pkgs[threadId]->input[15], pkgs[threadId]->input[16], pkgs[threadId]->input[17], pkgs[threadId]->input[18], pkgs[threadId]->input[19]);
            //printf("Thread id  %08x \n",pkgs[threadId]->input[19]);
            scrypt(     &(pkgs[threadId]->ctx), \
                        pkgs[threadId]->input, \
                        20, \
                        pkgs[threadId]->output, \
                        pkgs[threadId]->mem, \
                        pkgs[threadId]->salt, \
                        pkgs[threadId]->hmac_out, \
                        pkgs[threadId]->khash, \
                        pkgs[threadId]->in_ihash, \
                        pkgs[threadId]->in_2_ihash, \
                        pkgs[threadId]->ihash, \
                        pkgs[threadId]->in_ohash, \
                        pkgs[threadId]->pbkdf2_rm_out\
                    );
            // printf("!!!!Hello\n");
            // if(threadId == NUMTHREAD-1){
            //     // printf("i: %d\n", i);
            // }
            if(threadId == NUMTHREAD-1 && i==MAXNONCE/NUMTHREAD-1){
                 printf("Thread ID (%d) = %08x%08x%08x%08x%08x%08x%08x%08x\n", threadId, pkgs[threadId]->output[0],pkgs[threadId]->output[1],pkgs[threadId]->output[2],pkgs[threadId]->output[3],pkgs[threadId]->output[4],pkgs[threadId]->output[5],pkgs[threadId]->output[6],pkgs[threadId]->output[7]);
            }

            endian_cvt(&pkgs[threadId]->input[19]);
            pkgs[threadId]->input[19]++;
            // printf("Thread id %d: %u maxnonce: %u\n", threadId, pkgs[threadId]->input[19], MAXNONCE/NUMTHREAD);
            endian_cvt(&pkgs[threadId]->input[19]);
        }
        // printf("[%d]\n",\
        // threadId);
        
        // words_to_hex_string(out[threadId], 8, hex_str, 64);
        // printf("Thread ID: %d, %s", threadId, hex_str);
        // print_words_inline(&out[threadId], 8);
        

    }

}

int main()
{
  
    
    // static char test_scrypt_in[] =     "0000002056efd1943684c1fdc247d4759cc43b29afa1cac7ad14579de5f6abcbc6bdf448ee3de4c7b45e9496ab41ecde73d1a299ddbcc7a81aa52776e6c067e214233af097b6885c97df011aa004090e";
    // static char test_scrypt_in[] =     "0000002056efd1943684c1fdc247d4759cc43b29afa1cac7ad14579de5f6abcbc6bdf448ee3de4c7b45e9496ab41ecde73d1a299ddbcc7a81aa52776e6c067e214233af097b6885c97df011a";

    char ver[]="20000000";
    char prev_block[]="48f4bdc6cbabf6e59d5714adc7caa1af293bc49c75d447c2fdc1843694d1ef56";
    char mrkl_root[]="f03a2314e267c0e67627a51aa8c7bcdd99a2d173deec41ab96945eb4c7e43dee";
    char time[9];
    char bits[9];

    
    little_endian(ver, sizeof(ver) - 1);
    little_endian(prev_block, sizeof(prev_block) - 1);
    little_endian(mrkl_root, sizeof(mrkl_root) - 1);
    


    // Get time

    struct tm t;
    time_t t_of_day;

    t.tm_year = 2019-1900;  // Year - 1900
    t.tm_mon = 3-1;           // Month, where 1 = jan
    t.tm_mday = 13;          // Day of the month
    t.tm_hour = 7+9;
    t.tm_min = 51;
    t.tm_sec = 51;
    t.tm_isdst = -1;        // Is DST on? 1 = yes, 0 = no, -1 = unknown
    t_of_day = mktime(&t);
    
    WORD *wtime = new WORD(t_of_day);
    endian_cvt_host(wtime);
    word_to_hex_eight_host(*wtime, time, 8);

    word_to_hex_eight_host(436330391, bits, 8); // bits -- input
    little_endian(bits, 8);
    char test_scrypt_in[153];
   

  
    int in_index = 0;
    for(int i = 0; i < sizeof(ver)-1; i++){
        test_scrypt_in[i]=ver[i];
    }
    in_index += sizeof(ver)-1;
    for(int i = 0; i < sizeof(prev_block); i++){
        test_scrypt_in[in_index+i] = prev_block[i];
    }
    in_index += sizeof(prev_block)-1;
    for(int i = 0; i < sizeof(mrkl_root); i++){
        test_scrypt_in[in_index+i] = mrkl_root[i];
    }
    in_index += sizeof(mrkl_root)-1;
    for(int i = 0; i < sizeof(time); i++){
        test_scrypt_in[in_index+i] = time[i];
    }
    in_index += sizeof(time)-1;
    for(int i = 0; i < sizeof(bits); i++){
        test_scrypt_in[in_index+i] = bits[i];
    }
    WORD *nonce = (WORD*) malloc(sizeof(WORD));
    // WORD *nonce = new WORD(235472032);
    
    // WORD test_scrypt_out_w[8][NUMTHREAD];
    
    
    SCRYPT_PKG **pkgs = (SCRYPT_PKG**)malloc(NUMTHREAD*sizeof(SCRYPT_PKG*));

    // for(int k = 0; k<NUMTHREAD; k++){
    //     hex_string_to_words_host(test_scrypt_in, sizeof(test_scrypt_in), &test_scrypt_in_w[20*k]);
    //     *nonce = STARTATNONCE + (k * (MAXNONCE/NUMTHREAD));
    //     endian_cvt_host(nonce);
    //     test_scrypt_in_w[20*k+19] = *nonce;
        
    // }
    for(int k = 0; k<NUMTHREAD; k++){
        pkgs[k] = (SCRYPT_PKG*)malloc(sizeof(SCRYPT_PKG));
        hex_string_to_words_host(test_scrypt_in, sizeof(test_scrypt_in), pkgs[k]->input);
        *nonce = STARTATNONCE + (k * (MAXNONCE/NUMTHREAD));
        endian_cvt_host(nonce);
        pkgs[k]->input[19] = *nonce;
        // printf("%d \n", pkgs[k]);
        // printf("%08x", pkgs[k]->input);
        // print_words_inline_host(pkgs[k]->input, 20);
    }
   
    uint32_t threadsPerBlock = BLOCKSIZE;
    uint32_t blocksPerGrid =(threadsPerBlock + MAXNONCE - 1) / threadsPerBlock;
    // Add vectors in parallel.
    hipError_t cudaStatus = scryptWithCuda(pkgs, blocksPerGrid, threadsPerBlock);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "scryptWithCuda failed!\n");
        return 1;
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

SCRYPT_PKG* pkg_init(WORD* indata){
    SCRYPT_PKG* pkg;
    hipMallocManaged(&pkg, sizeof(SCRYPT_PKG));
    for (int j = 0; j < 20; j++){
        pkg->input[j] = indata[j];
        // printf("%08x\n", pkg->input[j]);
    }
    return pkg;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t scryptWithCuda(SCRYPT_PKG **pkgs, int block_per_grid, int thread_per_block)
{
    SCRYPT_PKG **pkgs_dev;
    hipError_t cudaStatus;
    // SCRYPT_PKG *pkg_pointer;
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMallocManaged((void**)&pkgs_dev, NUMTHREAD * sizeof(SCRYPT_PKG*));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    for(int i = 0; i<NUMTHREAD; i++){
        
        // Copy input vectors from host memory to GPU buffers.

        pkgs_dev[i] = pkg_init(pkgs[i]->input);
        // print_words_inline_host(pkgs[i]->input, 20);
        // printf("    %08x\n", pkgs_dev[i]);
        // cudaStatus = cudaMemcpy(pkgs_dev[i]->input, pkgs[i]->input, 20*sizeof(WORD), cudaMemcpyHostToDevice);
        // if (cudaStatus != cudaSuccess) {
        //     fprintf(stderr, "cudaMemcpy failed!");
        //     goto Error;
        // }
    }

    // Launch a kernel on the GPU with one thread for each element.

    scryptCuda<<<block_per_grid, thread_per_block>>>(pkgs_dev);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "scryptCuda launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching scryptCuda!\n", cudaStatus);
        goto Error;
    }

    // // Copy output vector from GPU buffer to host memory.

    // cudaStatus = cudaMemcpy(out, dev_out, NUMTHREAD * 8 * sizeof(WORD), cudaMemcpyDeviceToHost);
    // if (cudaStatus != cudaSuccess) {
    //     fprintf(stderr, "cudaMemcpy launch failed: %s\n", cudaGetErrorString(cudaStatus));
    //     goto Error;
    // }


Error:
    
    hipFree(pkgs_dev);
    
    return cudaStatus;
}

