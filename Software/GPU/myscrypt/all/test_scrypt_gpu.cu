#include "hip/hip_runtime.h"
// cd /home/hork/cuda-workspace/CudaSHA256/Debug/files
// time ~/Dropbox/FIIT/APS/Projekt/CpuSHA256/a.out -f ../file-list
// time ../CudaSHA256 -f ../file-list


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <memory.h>
#include <iostream>

#include <dirent.h>
#include <ctype.h>
#include <time.h>
#include "sha256.h"

#define N 10
#define M 100
#define MAXLOOP M/N
//#define N 6


#define checkCudaErrors(x) \
{ \
    hipGetLastError(); \
    x; \
    hipError_t err = hipGetLastError(); \
    if (err != hipSuccess) \
        printf("GPU: hipError_t %d (%s)\n", err, hipGetErrorString(err)); \
}

// datatypes -----------------------------------------------------------------------
#ifndef DATATYPES
#define DATATYPES
// Data types

typedef unsigned char BYTE; // 8-bit byte
typedef unsigned int  WORD; // 32-bit word
#endif

// sha256 ----------------------------------------------------------------------------------------

#ifndef PBKDF2
#define PBKDF2
#include <iostream>
#include "sha256.h"
#define ipad_elm 0x36363636
#define opad_elm 0x5c5c5c5c
#define SUM(a,b) (a+b) & 0xffffffff

// #define SALSA_MIX(destination ,a1, a2, b) (destination ^ (((SUM(a1,a2) << b) & 0xffffffff) | ((SUM(a1,a2) >> (32-b))&0xffffffff)))
#define SALSA_MIX(destination ,a1, a2, b) (destination ^ (ROTLEFT(SUM(a1,a2),b)))



const WORD IPAD[8] = {ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm}; // 256-bit 363636...36   
const WORD OPAD[8] = {opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm}; // 256-bit 5c5c5c...5c

// Function in scrypt
WORD* hmac(SHA256_CTX *ctx, WORD *salt, unsigned long salt_len, WORD *message, unsigned long message_len);
WORD* pbkdf2(SHA256_CTX *ctx, WORD *block, unsigned long block_len, int dklenP);
WORD* pbkdf2_2nd(SHA256_CTX *ctx, WORD *rm_out, unsigned long rm_out_len, WORD *block, unsigned long block_len, int dklenP);
void salsa_round(WORD *x1, WORD *x2, WORD *x3, WORD *x4);
WORD * salsa20_8(WORD *x);
WORD * blockmix(WORD *block);
WORD * romix(WORD *block, int N);
WORD * scrypt(SHA256_CTX *ctx, WORD *block, unsigned long block_len, int dklenP1, int N, int dklenP2);
#endif

static const WORD k[64] = {0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5, \
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174, \
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da, \
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967, \
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85, \
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070, \
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3, \
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2};
 
 // Create init state for SHA-256
 void sha256_init(SHA256_CTX *ctx)
 {
     ctx->datalen = 0;
     ctx->bitlen = 0;
     ctx->state[0] = h0;
     ctx->state[1] = h1;
     ctx->state[2] = h2;
     ctx->state[3] = h3;
     ctx->state[4] = h4;
     ctx->state[5] = h5;
     ctx->state[6] = h6;
     ctx->state[7] = h7;
 }
 
 
 void sha256_transform(SHA256_CTX *ctx, const BYTE data[])
 {
 // m is W in hardware design
   WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];
 // Calculate the first 16 m elements.
   for (i = 0, j = 0; i < 16; ++i, j += 4)
     m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);
 // Calculate the remain elements.
   for ( ; i < 64; ++i)
     m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];
 // update the new value of state after each block
   a = ctx->state[0];
   b = ctx->state[1];
   c = ctx->state[2];
   d = ctx->state[3];
   e = ctx->state[4];
   f = ctx->state[5];
   g = ctx->state[6];
   h = ctx->state[7];
 // process 64 rounds
   for (i = 0; i < 64; ++i) {
     t1 = h + EP1(e) + CH(e,f,g) + k[i] + m[i];
     t2 = EP0(a) + MAJ(a,b,c);
     h = g;
     g = f;
     f = e;
     e = d + t1;
     d = c;
     c = b;
     b = a;
     a = t1 + t2;
   }
 
   ctx->state[0] += a;
   ctx->state[1] += b;
   ctx->state[2] += c;
   ctx->state[3] += d;
   ctx->state[4] += e;
   ctx->state[5] += f;
   ctx->state[6] += g;
   ctx->state[7] += h;
 }
 // the total length of the message has to be specified
 void sha256_update(SHA256_CTX *ctx, const BYTE data[], size_t len)
 {
     WORD i;
 
     for (i = 0; i < len; ++i){
         ctx->data[ctx->datalen] = data[i];      // Pad data (message) for each 512-block in --> transform
         ctx->datalen++;
         // after browse for 64 bytes (512-bit block) -> transform the block.
         if(ctx->datalen == 64){
             sha256_transform(ctx, ctx->data);
             ctx->bitlen += 512; // increase the bit length by 512
             ctx->datalen = 0;
         }
     }
 }
 // this function processes for the last block -> after all real data is browsed 
 void sha256_final(SHA256_CTX *ctx, WORD *hash){
     WORD i;
 // padding is processed from here
     i = ctx->datalen;
     if (ctx->datalen < 56){
         // add byte 0x80 at the first if the datalength is lower than 56
         ctx->data[i++] = 0x80;
         // pad the zero bytes until the byte 56th
         while (i<56)
         {
             ctx->data[i++]=0x00;
         }
     }
     else{
         // add byte at the first
         ctx->data[i++]=0x80;
         // pad zero bytes until the last block
         while (i<64){
             ctx->data[i++]=0x00;
         }
         // transform this block --> it's not the last block
         sha256_transform(ctx, ctx->data);
         // set 56 zero bytes from last_block[0:55]
         memset(ctx->data, 0, 56);
     }
 
     // Append to the padding the total message's length in bits and transform.
     ctx->bitlen += ctx->datalen * 8;
     ctx->data[63] = ctx->bitlen;
     ctx->data[62] = ctx->bitlen >> 8;
     ctx->data[61] = ctx->bitlen >> 16;
     ctx->data[60] = ctx->bitlen >> 24;
     ctx->data[59] = ctx->bitlen >> 32;
     ctx->data[58] = ctx->bitlen >> 40;
     ctx->data[57] = ctx->bitlen >> 48;
     ctx->data[56] = ctx->bitlen >> 56;
 // end padding
     sha256_transform(ctx, ctx->data);
 
     // Since this implementation uses little endian byte ordering and SHA uses big endian,
     // reverse all the bytes when copying the final state to the output hash.
 
 
         hash[0] = ctx->state[0];
         hash[1] = ctx->state[1];
         hash[2] = ctx->state[2];
         hash[3] = ctx->state[3];
         hash[4] = ctx->state[4];
         hash[5] = ctx->state[5];
         hash[6] = ctx->state[6];
         hash[7] = ctx->state[7]; 
  
 }
 
 char * sha256(SHA256_CTX *ctx, char hex_str_in[], unsigned long hex_str_len){
     
     unsigned long datalen = hex_str_len/2;
     BYTE *data=new BYTE[datalen]();
     // WORD hash_w[8]; //--> true
     // WORD *hash_w = (WORD*)malloc(sizeof(WORD)*8); //--> true
     WORD *hash_w = new WORD[64](); //--> false
     static char *out = new char[64]();
   
     hex_string_to_bytes(hex_str_in, hex_str_len, data);
     sha256_init(ctx);
 
     sha256_update(ctx, data, datalen);
 
     sha256_final(ctx,hash_w);
     words_to_hex_string(hash_w, 8, out, 64);
     return out;
 }
 
 void sha256_w(SHA256_CTX *ctx, char hex_str_in[], unsigned long hex_str_len, WORD *hash_w){
     
     unsigned long datalen = hex_str_len/2;
     BYTE *data=new BYTE[datalen]();
   
     hex_string_to_bytes(hex_str_in, hex_str_len, data);
     sha256_init(ctx);
  
     sha256_update(ctx, data, datalen);
 
     sha256_final(ctx, hash_w);
 }
 
 void sha256_in_bytes(SHA256_CTX *ctx, BYTE *bytes_in, unsigned long bytes_in_len, WORD *hash_w){
     sha256_init(ctx);
  
     sha256_update(ctx, bytes_in, bytes_in_len);
 
     sha256_final(ctx, hash_w);
 }
 
 void sha256_in_words(SHA256_CTX *ctx, WORD *words_in, unsigned long words_in_len, WORD *hash_w){
     unsigned bytes_in_len = words_in_len * 4;
     BYTE bytes_in[bytes_in_len];
 
     for (int i = 0; i<words_in_len; i++){
         bytes_in[4*i] = words_in[i] >> 24;
         bytes_in[4*i+1] = words_in[i] >> 16;
         bytes_in[4*i+2] = words_in[i] >> 8;
         bytes_in[4*i+3] = words_in[i];
     }
     
     sha256_init(ctx);
  
     sha256_update(ctx, bytes_in, bytes_in_len);
 
     sha256_final(ctx, hash_w);
 }


 // utils ------------------------------------------------------------------------------------------

 #ifndef UTILS
#define UTILS
#include "datatypes.h"
#include <iostream>
#include <algorithm>
#include <stddef.h>

BYTE hex_char_to_byte(char hex_char);
void hex_string_to_bytes(char hex_str_in[], unsigned long hex_str_len, BYTE bytes_out[]);
void half_byte_to_hex(BYTE half_byte_in, char hex);
void word_to_hex_eight(WORD word_in, char *hex_eight, unsigned long hex_eight_size);
void words_to_hex_string(WORD words_in[], unsigned long words_len, char hex_str[], unsigned long hex_str_len);
void hex_string_to_words(char hex_str_in[], unsigned long hex_str_len, WORD words_out[]);
void add_two_words_array_512_bit(WORD *a, WORD *b);
void print_words_inline(WORD *w, unsigned long w_len);
void print_words_multiline(WORD *w, unsigned long w_len);
void add_two_words_array_512_bit_with_carry(WORD *a, WORD *b);
void endian_cvt(WORD *w);
void endian_full(WORD *w, unsigned long w_len);
void little_endian(char *c, unsigned long w_len);
#endif

// ----------------------- Utils functions ------------------------
BYTE hex_char_to_byte(char hex_char){
    if(hex_char >= 'a' && hex_char <='f'){
        return hex_char - 'a' + 10;
    }
    else if(hex_char >='A' && hex_char <= 'F'){
        return hex_char - 'A' + 10;
    } else if (hex_char >='0' && hex_char <= '9')
    {
        return hex_char - '0';
    }
    return 0;
}
void hex_string_to_bytes(char hex_str_in[], unsigned long hex_str_len, BYTE bytes_out[]){
    for (int i = 0; i<hex_str_len-1; i+=2){
        bytes_out[i/2] = ((hex_char_to_byte(hex_str_in[i])) << 4) | (hex_char_to_byte(hex_str_in[i+1]));

    }
}

void hex_string_to_words(char hex_str_in[], unsigned long hex_str_len, WORD words_out[]){
    for (int i = 0; i<hex_str_len-1; i+=8){
        words_out[i/8] = (\
            hex_char_to_byte(hex_str_in[i])<<28|\
            (hex_char_to_byte(hex_str_in[i+1])<<24 & 0x0f000000)|\
            (hex_char_to_byte(hex_str_in[i+2])<<20 & 0x00f00000)|\
            (hex_char_to_byte(hex_str_in[i+3])<<16 & 0x000f0000)|\
            (hex_char_to_byte(hex_str_in[i+4])<<12 & 0x0000f000)|\
            (hex_char_to_byte(hex_str_in[i+5])<<8  & 0x00000f00)|\
            (hex_char_to_byte(hex_str_in[i+6])<<4  & 0x000000f0)|\
            (hex_char_to_byte(hex_str_in[i+7])     & 0x0000000f)\
        );
        // printf("%08x  %d\n", words_out[i/8], i/8);
    }
}
void half_byte_to_hex(BYTE half_byte_in, char *hex){
    BYTE half_byte_conv = half_byte_in & 0x0f;
    if(half_byte_conv<16){
        if (half_byte_conv>=10){
            *hex = 'a'+ half_byte_conv - 10;
            // printf("%c\n", *hex);
            return;
        }
        else if(half_byte_conv>=0){
            *hex = '0' + half_byte_conv;
            // printf("%c\n", *hex);
            return;
        }
    }
    printf("The half byte must be in range of [0:15]\n");
}

void word_to_hex_eight(WORD word_in, char *hex_eight, unsigned long hex_eight_size){
    if(hex_eight_size==8){
        half_byte_to_hex(word_in>>28, &hex_eight[0]);
        half_byte_to_hex(word_in>>24, &hex_eight[1]);
        half_byte_to_hex(word_in>>20, &hex_eight[2]);
        half_byte_to_hex(word_in>>16, &hex_eight[3]);
        half_byte_to_hex(word_in>>12, &hex_eight[4]);
        half_byte_to_hex(word_in>>8, &hex_eight[5]);
        half_byte_to_hex(word_in>>4, &hex_eight[6]);
        half_byte_to_hex(word_in, &hex_eight[7]);
        // printf("%c", hex_eight[0]);
        // printf("%d", word_in>>24);
        return;
    }
    printf("The hex_pair must have the length of two characters: %d\n", (int)hex_eight_size);
}

void words_to_hex_string(WORD *words_in, unsigned long words_len, char hex_str[], unsigned long hex_str_len){
    char hex_eight[8];
    if(hex_str_len == 8*words_len){
        for (int i = 0; i<words_len; ++i){
            // printf("\n w: %08x", words_in[i]);
            word_to_hex_eight(words_in[i], hex_eight, sizeof(hex_eight));
            hex_str[8*i] = hex_eight[0];
            hex_str[8*i+1] = hex_eight[1];
            hex_str[8*i+2] = hex_eight[2];
            hex_str[8*i+3] = hex_eight[3];
            hex_str[8*i+4] = hex_eight[4];
            hex_str[8*i+5] = hex_eight[5];
            hex_str[8*i+6] = hex_eight[6];
            hex_str[8*i+7] = hex_eight[7];
            // printf("%c \n", hex_eight[7]);
        }
        // printf("\n%s", hex_str);
        return;
    }
    printf("The hex_string must have the lenght of 4*bytes_len: %d\n", (int)hex_str_len);
}

void add_two_words_array_512_bit(WORD *a, WORD *b){
    
    for (int i = 15; i>=0; i--){

        a[i] += b[i];

    }
}

void add_two_words_array_512_bit_with_carry(WORD *a, WORD *b){
    WORD sum = 0;
    WORD sum1 = 0;
    
    for (int i = 15; i>=0; i--){

        sum = ((a[i]&0x0000ffff)+(b[i]&0x0000ffff)+(sum1>>16));
        sum1 = ((a[i]>>16)+(b[i]>>16)+(sum>>16));
        a[i]= (sum & 0x0000ffff) + (sum1<<16);

    }
}

void print_words_inline(WORD *w, unsigned long w_len){
    printf("\n");
    for (int i = 0; i< w_len; i++){
        printf("%08x", w[i]);
    }
    printf("\n");
}

void print_words_multiline(WORD *w, unsigned long w_len){
    printf("\n");
    for (int i = 0; i< w_len; i++){
        printf("%08x\n", w[i]);
    }
    printf("\n");
}

void endian_cvt(WORD *w){
    WORD out;

    out = (*w>>24)|((*w>>8)&0x0000ff00)|((*w<<8)&0x00ff0000)|(*w<<24);

    *w = out;
}

void endian_full(WORD *w, unsigned long w_len){
    for (int i = 0; i < w_len; i++)
    {
        endian_cvt(&w[i]);
    }
}
void little_endian(char *c, unsigned long w_len){
    char dc[w_len];

    for (int i = 0; i< w_len; i+=2){
        dc[w_len-2-i] = c[i];
        dc[w_len-1-i] = c[i+1];   
    }
    for (int i = 0; i< w_len; i++){
        c[i] = dc[i];

    }
    c[w_len] = '\0';
    
}

// scrypt --------------------------------------------------------------------------------------------------------------------------

#ifndef PBKDF2
#define PBKDF2
#include <iostream>
#include "sha256.h"
#define ipad_elm 0x36363636
#define opad_elm 0x5c5c5c5c
#define SUM(a,b) (a+b) & 0xffffffff

// #define SALSA_MIX(destination ,a1, a2, b) (destination ^ (((SUM(a1,a2) << b) & 0xffffffff) | ((SUM(a1,a2) >> (32-b))&0xffffffff)))
#define SALSA_MIX(destination ,a1, a2, b) (destination ^ (ROTLEFT(SUM(a1,a2),b)))



const WORD IPAD[8] = {ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm, ipad_elm}; // 256-bit 363636...36   
const WORD OPAD[8] = {opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm, opad_elm}; // 256-bit 5c5c5c...5c

// Function in scrypt
WORD* hmac(SHA256_CTX *ctx, WORD *salt, unsigned long salt_len, WORD *message, unsigned long message_len);
WORD* pbkdf2(SHA256_CTX *ctx, WORD *block, unsigned long block_len, int dklenP);
WORD* pbkdf2_2nd(SHA256_CTX *ctx, WORD *rm_out, unsigned long rm_out_len, WORD *block, unsigned long block_len, int dklenP);
void salsa_round(WORD *x1, WORD *x2, WORD *x3, WORD *x4);
WORD * salsa20_8(WORD *x);
WORD * blockmix(WORD *block);
WORD * romix(WORD *block, int N);
WORD * scrypt(SHA256_CTX *ctx, WORD *block, unsigned long block_len, int dklenP1, int N, int dklenP2);
#endif

WORD* hmac(SHA256_CTX *ctx, WORD *salt, unsigned long salt_len, WORD *message, unsigned long message_len){
    WORD *khash = (WORD*) malloc(sizeof(WORD)*8);
    sha256_in_words(ctx, message, message_len, khash);
    // for(int i=0;i<8; i++){
    //     printf("%08x", khash[i]);
    // }
    WORD ixor[16] = {\
        IPAD[0]^khash[0],\
        IPAD[1]^khash[1],\
        IPAD[2]^khash[2],\
        IPAD[3]^khash[3],\
        IPAD[4]^khash[4],\
        IPAD[5]^khash[5],\
        IPAD[6]^khash[6],\
        IPAD[7]^khash[7],\
        IPAD[0],\
        IPAD[1],\
        IPAD[2],\
        IPAD[3],\
        IPAD[4],\
        IPAD[5],\
        IPAD[6],\
        IPAD[7],\
    };

    WORD oxor[16] = {\
        OPAD[0]^khash[0],\
        OPAD[1]^khash[1],\
        OPAD[2]^khash[2],\
        OPAD[3]^khash[3],\
        OPAD[4]^khash[4],\
        OPAD[5]^khash[5],\
        OPAD[6]^khash[6],\
        OPAD[7]^khash[7],\
        OPAD[0],\
        OPAD[1],\
        OPAD[2],\
        OPAD[3],\
        OPAD[4],\
        OPAD[5],\
        OPAD[6],\
        OPAD[7],\
    };

    WORD in_ihash[sizeof(ixor)/sizeof(WORD)+salt_len];
    int i;
    for(i = 0; i<sizeof(ixor)/sizeof(WORD); i++){
        in_ihash[i] = ixor[i];
    }
   
    for(;i<sizeof(ixor)/sizeof(WORD)+salt_len; i++){
        in_ihash[i] = salt[i-sizeof(ixor)/sizeof(WORD)];
    }

    WORD ihash[8];
    sha256_in_words(ctx, in_ihash, sizeof(in_ihash)/sizeof(WORD), ihash);

    WORD in_ohash[sizeof(oxor)/sizeof(WORD)+sizeof(ihash)/sizeof(WORD)];
    for(i = 0; i<sizeof(oxor)/sizeof(WORD); i++){
        in_ohash[i] = oxor[i];
    }
    for(;i<sizeof(ixor)/sizeof(WORD)+salt_len; i++){
        in_ohash[i] = ihash[i-sizeof(oxor)/sizeof(WORD)];
    }


    static WORD ohash[8];
    sha256_in_words(ctx, in_ohash, sizeof(in_ohash)/sizeof(WORD), ohash);
    return ohash;
}

WORD* pbkdf2(SHA256_CTX *ctx, WORD *block, unsigned long block_len, int dklenP){
    int num_loop = 1024/dklenP;
    WORD salt[block_len+1];
    WORD *hmac_out;
    // int hmac_out_len = 8;
    static WORD *pbkdf2_out = new WORD[num_loop*8]();
    
    for(int i = 0; i<block_len; i++){
        salt[i]=block[i];
    }
    for (int i = 1; i <= num_loop; i++)
    {
        salt[block_len] = i;
        hmac_out = hmac(ctx, salt, block_len+1, block, block_len);
        for(int j = 0; j<8; j++){
            pbkdf2_out[(i-1)*8+j] = hmac_out[j];
        }
    }
    return pbkdf2_out;    
}

WORD* pbkdf2_2nd(SHA256_CTX *ctx, WORD *rm_out, unsigned long rm_out_len, WORD *block, unsigned long block_len, int dklenP){
    int num_loop = 1024/dklenP;
    WORD salt[rm_out_len+1];
    WORD *hmac_out;
    // int hmac_out_len = 8;
    static WORD *pbkdf2_out = new WORD[num_loop*8]();
    
    for(int i = 0; i<rm_out_len; i++){
        salt[i]=rm_out[i];
    }
    for (int i = 1; i <= num_loop; i++)
    {
        salt[rm_out_len] = i;
        hmac_out = hmac(ctx, salt, rm_out_len+1, block, block_len);
        for(int j = 0; j<8; j++){
            pbkdf2_out[(i-1)*8+j] = hmac_out[j];
        }
    }
    return pbkdf2_out; 
}
void salsa_round(WORD *x1, WORD *x2, WORD *x3, WORD *x4){
    *x1 = SALSA_MIX(*x1, *x4, *x3, 7);
    *x2 = SALSA_MIX(*x2, *x1, *x4, 9);
    *x3 = SALSA_MIX(*x3, *x2, *x1, 13);
    *x4 = SALSA_MIX(*x4, *x3, *x2, 18);
}

WORD * salsa20_8(WORD *x){
    static WORD out[16];
    for(int i = 0; i<4; i++){
        salsa_round(&x[4], &x[8], &x[12], &x[0]);
        salsa_round(&x[9], &x[13], &x[1], &x[5]);
        salsa_round(&x[14], &x[2], &x[6], &x[10]);
        salsa_round(&x[3], &x[7], &x[11], &x[15]);
        salsa_round(&x[1], &x[2], &x[3], &x[0]);
        salsa_round(&x[6], &x[7], &x[4], &x[5]);
        salsa_round(&x[11], &x[8], &x[9], &x[10]);
        salsa_round(&x[12], &x[13], &x[14], &x[15]);
    }
    for(int i=0; i<16; i++){
        out[i] = x[i];
    }
    return out;
}

WORD * blockmix(WORD *block){
    WORD x_arr[16];
    WORD x_arr_cpy[16];
    static WORD *out = new WORD[32]();
    for (int i = 0; i < 16; i++){
        x_arr[i] = block[i];
    }
    
    for (int i = 0; i<2; i++){
        for (int j = 0; j < 16; j++){
            x_arr_cpy[j] = x_arr[j] ^ block[j+16];
            x_arr[j] ^= block[j+16];
        }
        add_two_words_array_512_bit(x_arr, salsa20_8(x_arr_cpy));
        for (int j = 0; j < 16; j++){
            out[(16*i)+j] = x_arr[j]; 
        }
    }
    return out;
}

WORD * romix(WORD *block, int N){
    WORD mem[1024][32];
    static WORD *out = new WORD[32]();
    int j;
    for (int i = 0; i<N; i++){
        for (j = 0; j < 32; j++){
            mem[i][j] = block[j];
        }
        block = blockmix(block);
    }
    for (int i = 0; i<N; i++){
        j = (block[16] & 0x000003ff);
 
        for (int k = 0; k<32; k++){
            block[k] ^= mem[j][k];
        }
        block = blockmix(block);

    }
    out = block;
    return out;
}

WORD * scrypt(SHA256_CTX *ctx, WORD *block, unsigned long block_len, int dklenP1, int N, int dklenP2){
    int pbkdf2_out_len_1 = 8*(1024/dklenP1);
    int pbkdf2_out_len_2 = 8*(1024/dklenP2);
    WORD *pbkdf2_1_out = new WORD[pbkdf2_out_len_1]();
    WORD *romix_out = new WORD[32]();
    static WORD *pbkdf2_2_out = new WORD[pbkdf2_out_len_2]();
    pbkdf2_1_out = pbkdf2(ctx, block, block_len, dklenP1);
  
    endian_full(pbkdf2_1_out, pbkdf2_out_len_1);
    romix_out = romix(pbkdf2_1_out, N);
    endian_full(romix_out, 32);
    pbkdf2_2_out = pbkdf2_2nd(ctx, romix_out, 32, block, block_len, dklenP2);
    return pbkdf2_2_out;
}

__device__ void scrypt_cuda(SHA256_CTX *ctx, WORD block[], unsigned long block_len, int dklenP1, int N, int dklenP2, WORD hash_out[])
{
    hash_out = scrypt(ctx, block, block_len, dklenP1, N, dklenP2);
}

__global__ void scrypt_top_cuda(uint32_t max_loop) {

    uint32_t index  = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x;
    uint32_t j;
    for (j = index; j < N; j += stride){
        SHA256_CTX *ctx = new SHA256_CTX();
        char ver[]="20000000";
        char prev_block[]="48f4bdc6cbabf6e59d5714adc7caa1af293bc49c75d447c2fdc1843694d1ef56";
        char mrkl_root[]="f03a2314e267c0e67627a51aa8c7bcdd99a2d173deec41ab96945eb4c7e43dee";
        char time[9];
        char bits[9];
        little_endian(ver, sizeof(ver) - 1);
        little_endian(prev_block, sizeof(prev_block) - 1);
        little_endian(mrkl_root, sizeof(mrkl_root) - 1);
        
        // Get time

        struct tm t;
        time_t t_of_day;

        t.tm_year = 2019-1900;  // Year - 1900
        t.tm_mon = 3-1;           // Month, where 1 = jan
        t.tm_mday = 13;          // Day of the month
        t.tm_hour = 7+9;
        t.tm_min = 51;
        t.tm_sec = 51;
        t.tm_isdst = -1;        // Is DST on? 1 = yes, 0 = no, -1 = unknown
        t_of_day = mktime(&t);
        
        WORD *wtime = new WORD(t_of_day);
        endian_cvt(wtime);
        word_to_hex_eight(*wtime, time, 8);

        word_to_hex_eight(436330391, bits, 8); // bits -- input
        little_endian(bits, 8);
        char test_scrypt_in[153];


        
        int in_index = 0;
        WORD i;
        for( i = 0; i < sizeof(ver)-1; i++){
            test_scrypt_in[i]=ver[i];
        }
        in_index += sizeof(ver)-1;
        for( i = 0; i < sizeof(prev_block); i++){
            test_scrypt_in[in_index+i] = prev_block[i];
        }
        in_index += sizeof(prev_block)-1;
        for( i = 0; i < sizeof(mrkl_root); i++){
            test_scrypt_in[in_index+i] = mrkl_root[i];
        }
        in_index += sizeof(mrkl_root)-1;
        for( i = 0; i < sizeof(time); i++){
            test_scrypt_in[in_index+i] = time[i];
        }
        in_index += sizeof(time)-1;
        for( i = 0; i < sizeof(bits); i++){
            test_scrypt_in[in_index+i] = bits[i];
        }


        WORD *test_scrypt_out_w = new WORD[8]();
        char *test_scrypt_out = new char[32*8](); 
        WORD test_scrypt_in_w[20];

        for (i = j*max_loop; i<(j+1)*max_loop; i++){
            hex_string_to_words(test_scrypt_in, sizeof(test_scrypt_in), test_scrypt_in_w);
            test_scrypt_in_w[19] = i;
            endian_cvt(&test_scrypt_in_w[19]);
            scrypt_cuda(ctx, test_scrypt_in_w, 20, 256, 1024, 1024, test_scrypt_out_w);
            if(i==(index+1)*max_loop-1){
                printf("\nThread id: %d, nonce: %d\n", index, i);
            }
        }
    }

}


int main(void)
{

    
	int GPU_N;
	checkCudaErrors(hipGetDeviceCount(&GPU_N));
	printf("CUDA-capable device count: %d\n", GPU_N);
	checkCudaErrors(hipSetDevice(GPU_N-1));




    uint32_t blockSize = 256;
    uint32_t numBlocks = (N + blockSize - 1) / blockSize;
    // uint32_t *max_loop_cpu = (uint32_t *)malloc(sizeof(uint32_t));
    // *max_loop_cpu = M;
	// checkCudaErrors(hipMallocManaged(&max_loop_gpu, sizeof(uint32_t)));
    // hipMemcpy(max_loop_gpu, max_loop_cpu, hipMemcpyHostToDevice);

	
    scrypt_top_cuda <<<numBlocks, blockSize>>> (MAXLOOP);

	hipDeviceReset();
	
	return 0;
}	
	
